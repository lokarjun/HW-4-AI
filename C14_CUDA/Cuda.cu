#include "hip/hip_runtime.h"
%%writefile fibonacci.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <fstream>

#define N (1 << 18)

__global__ void fibonacci(unsigned long long *fib) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i == 0) fib[0] = 0;
    else if (i == 1) fib[1] = 1;
    else if (i < N) {
        unsigned long long a = 0, b = 1, c;
        for (int j = 2; j <= i; j++) {
            c = a + b;
            a = b;
            b = c;
        }
        fib[i] = b;
    }
}

int main() {
    auto wall_start = std::chrono::high_resolution_clock::now();

    unsigned long long *d_fib, *h_fib;
    size_t size = N * sizeof(unsigned long long);

    h_fib = (unsigned long long *)malloc(size);
    hipMalloc(&d_fib, size);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    fibonacci<<<blocksPerGrid, threadsPerBlock>>>(d_fib);

    hipMemcpy(h_fib, d_fib, size, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float kernel_time = 0;
    hipEventElapsedTime(&kernel_time, start, stop);

    hipFree(d_fib);
    free(h_fib);

    auto wall_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> total_time = wall_end - wall_start;

    // Save total time to file
    std::ofstream fout("gpu_time.txt");
    fout << total_time.count();
    fout.close();

    return 0;
}
