#include "hip/hip_runtime.h"
%%writefile q_learning_grid.cu
#include <iostream>
#include <fstream>
#include <hiprand/hiprand_kernel.h>

#define BOARD_ROWS 5
#define BOARD_COLS 5
#define ACTIONS 4
#define NUM_SIZES 3

// Reward constants
#define REWARD_WIN 1
#define REWARD_HOLE -5
#define REWARD_DEFAULT -1

__device__ int holes[][2] = {{1,0},{3,1},{4,2},{1,3}};
__device__ bool isHole(int x, int y) {
    for (int i = 0; i < 4; i++) {
        if (holes[i][0] == x && holes[i][1] == y) return true;
    }
    return false;
}

__device__ int getReward(int x, int y) {
    if (x == 4 && y == 4) return REWARD_WIN;
    if (isHole(x, y)) return REWARD_HOLE;
    return REWARD_DEFAULT;
}

__global__ void setupKernel(hiprandState *state) {
    int id = threadIdx.x;
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void q_learning(float *Q, hiprandState *globalState, int EPISODES) {
    int tid = threadIdx.x;
    if (tid != 0) return;

    hiprandState localState = globalState[tid];
    float alpha = 0.5f, gamma = 0.9f, epsilon = 0.1f;

    for (int ep = 0; ep < EPISODES; ep++) {
        int x = 0, y = 0;
        bool isEnd = false;

        while (!isEnd) {
            int action;
            float r = hiprand_uniform(&localState);
            if (r < epsilon) {
                action = hiprand(&localState) % ACTIONS;
            } else {
                float maxQ = -1e9;
                for (int a = 0; a < ACTIONS; a++) {
                    float q = Q[(x * BOARD_COLS + y) * ACTIONS + a];
                    if (q > maxQ) {
                        maxQ = q;
                        action = a;
                    }
                }
            }

            int new_x = x, new_y = y;
            if (action == 0 && x > 0) new_x--;
            else if (action == 1 && x < BOARD_ROWS - 1) new_x++;
            else if (action == 2 && y > 0) new_y--;
            else if (action == 3 && y < BOARD_COLS - 1) new_y++;

            int reward = getReward(new_x, new_y);
            bool done = (new_x == 4 && new_y == 4) || isHole(new_x, new_y);

            float maxQNext = -1e9;
            for (int a = 0; a < ACTIONS; a++) {
                float q = Q[(new_x * BOARD_COLS + new_y) * ACTIONS + a];
                if (q > maxQNext) maxQNext = q;
            }

            int idx = (x * BOARD_COLS + y) * ACTIONS + action;
            Q[idx] = (1 - alpha) * Q[idx] + alpha * (reward + gamma * maxQNext);

            x = new_x;
            y = new_y;
            isEnd = done;
        }
    }

    globalState[tid] = localState;
}

int main() {
    int episode_sizes[NUM_SIZES] = {1000, 5000, 10000};

    std::ofstream logFile("gpu_time_log.txt");

    for (int test = 0; test < NUM_SIZES; test++) {
        int EPISODES = episode_sizes[test];

        float *d_Q;
        hiprandState *devStates;
        size_t size = BOARD_ROWS * BOARD_COLS * ACTIONS * sizeof(float);

        hipMalloc((void**)&d_Q, size);
        hipMemset(d_Q, 0, size);
        hipMalloc((void**)&devStates, sizeof(hiprandState));

        setupKernel<<<1, 1>>>(devStates);
        hipDeviceSynchronize();

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        q_learning<<<1, 1>>>(d_Q, devStates, EPISODES);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);

        std::cout << "Episodes: " << EPISODES << ", Time: " << ms / 1000.0f << " seconds\n";
        logFile << EPISODES << "," << ms / 1000.0f << "\n";

        hipFree(d_Q);
        hipFree(devStates);
    }

    logFile.close();
    return 0;
}
